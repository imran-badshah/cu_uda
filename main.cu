
#include <hip/hip_runtime.h>
#include <stdio.h>

// __global__ : declaration specifier (aka declspec): a C-language construct => CUDA knows this is a kernel and not CPU code
__global__ void cube(float *d_out, float *d_in) {
    int idx = threadIdx.x; //(dim3: a C-Struct with .x .y and .z)tells each thread its index within a block
    float f = d_in[idx];
    d_out[idx] = f * f *f;
}

int main(int argc, char **argv) {
    const int ARRAY_SIZE = 96;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // generate the input array on the host
    float h_in[ARRAY_SIZE];
    for (int i=0; i<ARRAY_SIZE; i++) {
        h_in[i] = float(i);
    }
    float h_out[ARRAY_SIZE];

    // declare GPU memory pointers
    float *d_in;
    float *d_out;

    // allocate GPU memory for the above pointers
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);

    // transfer the array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    // launch the kernel on one block of 96 threads | KERNEL <<<grid of blocks, blocks of threads>>>(...) | 1, 2 or 3D - dim3(x,y,z) dim3(w,1,1) == dim3(w) == w
    // kernel<<<dim3(bx,by,bz), dim3(tx,ty,tz), shmem>>>(...) | shmem = shared mem per block in bytes -> defaults to 0
    cube<<<1, ARRAY_SIZE>>>(d_out, d_in); // tells the CPU to launch on the GPU 96 copies of the kernel on 96 threads

    // copy back the result array to the CPU
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    // print out the resulting array
    for (int i=0; i<ARRAY_SIZE; i++) {
        printf("%f", h_out[i]);
        printf(((i % 4) != 3) ? "\t" : "\n");
    }

    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
